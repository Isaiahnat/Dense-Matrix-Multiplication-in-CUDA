#include "hip/hip_runtime.h"
// System Utilities
#include <iostream>
#include <cmath>
#include <assert.h>
#include <helper_string.h>

// CUDA Runtime
#include <hip/hip_runtime.h>
#include <hipblas.h>

// CUDA and CUBLAS helpers
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

// Define Block Size
#define BLOCK_SIZE 32


// Returns a matrix filled with garbage values
float * get_matrix(int dim) {

    // compute required memory for matrix
    int size = dim*dim*sizeof(float);
    
    // allocate memory for matrix
    float * matrix = (float *) malloc(size);

    return matrix;
}

// Returns matrix filled with random float values
// Matrix formatted in row major order
float * randomize_matrix(int dim) {

    // initialize empty matrix
    float * matrix = get_matrix(dim);

    // fill matrix with random floats
    for (int i = 0; i < dim*dim; i++) {

        // fill with values in the range [0.0, 1.0] to check decimal accuracy
        matrix[i] = rand()/(float)RAND_MAX;
    }

    return matrix;
}

// Sets up Device Memory for matrices
float * device_matrix(int dim) {

    // initialize pointer
    void * dev_ptr;

    // compute required memory for matrix
    int size = dim*dim*sizeof(float);

    // allocate device memory
    checkCudaErrors(hipMalloc(&dev_ptr, size));

    return (float *) dev_ptr;
}

// Copies matrix from host memory to device memory
void host_to_device(const void* host_ptr, void * dev_ptr, int dim) {

    // compute memory to be copied
    int size = dim*dim*sizeof(float);

    // copy memory to device
    checkCudaErrors(hipMemcpy(dev_ptr, host_ptr, size, hipMemcpyHostToDevice));

}

// Copies matrix from device memory to host memory
void device_to_host(const void * dev_ptr, void * host_ptr, int dim) {

    // compute memory to be copied
    int size = dim*dim*sizeof(float);

    // copy memory to host
    checkCudaErrors(hipMemcpy(host_ptr, dev_ptr, size, hipMemcpyDeviceToHost));

}

// Compares the result of our implementation of matmul to cublas
// Ensures difference between two entries is less than err
void compare_matrix(float * C_alg, float * C_cub, int size, float err) {

    // number of entries in matrix
    int num_entries = size * size;


    // check every entry
    for (int i = 0; i < num_entries; i++) {
        
        // error outside of acceptable range
        if (std::fabs(C_alg[i] - C_cub[i]) > err) {
            std::cout << "Failed: entry " << i <<" does not match" << std::endl;
            std::cout << "Alg entry: " << C_alg[i] << ", Cublas entry: " << C_cub[i] << std::endl;
            std::cout << "Difference: " << std::fabs(C_alg[i] - C_cub[i]) << std::endl;
            return;
        }
    }

    // matrices are identical to within the error margin
    std::cout << "Passed" << std::endl;
}

// Shared Memory Cache Blocking Matrix Multiplication Algorithm
// Assumes size is a multiple of BLOCK_SIZE
__global__ void mat_mul(const float * A, const float * B, float * C, int size) {

    // Initialize Scratch Space
    __shared__ float A_shared[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float B_shared[BLOCK_SIZE][BLOCK_SIZE];

    // Compute col index of result in C
    int col = blockIdx.x*blockDim.x + threadIdx.x;

    // Compute row index of result in C
    int row = blockIdx.y*blockDim.y + threadIdx.y;

    // Compute in memory index
    int index = row*size + col;

    float total = 0.0;


    for (int i = 0; i < size; i += BLOCK_SIZE) {

        // Each thread is responsible for loading a float
        // from global memory to shared memory


        // Index of global mem that thread loads from
        int A_index_global = row * size + threadIdx.x + i;
        int B_index_global = (i+threadIdx.y) * size + col;

        
        // Load into shared memory
        A_shared[threadIdx.y][threadIdx.x] = A[A_index_global];
        B_shared[threadIdx.y][threadIdx.x] = B[B_index_global];

        // Let threads load data into shared mem        
        __syncthreads();
        
        
        // Compute partial totals
        for (int j = 0; j < BLOCK_SIZE; j++) {

            total += A_shared[threadIdx.y][j] *
            B_shared[j][threadIdx.x];

        }

        // Let threads finish computing before loading next block
        __syncthreads();
    }

    // Store the result in device memory
    C[index] = total;
}




// Runs test comparing implementation of matmul to cublas
// for two square matrices of dim size x size
void run_test(int size) {

    // create matrices A and B and fill with random values
    float * A = randomize_matrix(size);
    float * B = randomize_matrix(size);

    // scratch space to hold end result of matmul
    float * C_cub = get_matrix(size);
    float * C_alg = get_matrix(size);

    // allocate device memory
    float * A_d = device_matrix(size);
    float * B_d = device_matrix(size);
    float * C_d_cub = device_matrix(size);
    float * C_d_alg = device_matrix(size);

    // set parameters
    float alpha = 1.0f;
    float beta = 0.0f;

    // set two dimensional grid, enough to cover matrix
    dim3 grid_dim(size/BLOCK_SIZE, size/BLOCK_SIZE, 1); 

    // set two dimensional thread block of 1024 threads
    dim3 block_dim(BLOCK_SIZE, BLOCK_SIZE, 1);
    
    // initialize cublas handle
    hipblasHandle_t handle;
    checkCudaErrors(hipblasCreate(&handle));

    // Copy A from host memory to device memory
    host_to_device(A, A_d, size);

    // Copy B from host memory to device memory
    host_to_device(B, B_d, size);

    // Initialize events for timing diagnostics'
    hipEvent_t start_cub;
    hipEvent_t stop_cub;
    hipEvent_t start_alg;
    hipEvent_t stop_alg;

    // Create CUDA events that we'll use for timing
    checkCudaErrors(hipEventCreate(&start_cub));
    checkCudaErrors(hipEventCreate(&stop_cub));
    checkCudaErrors(hipEventCreate(&start_alg));
    checkCudaErrors(hipEventCreate(&stop_alg));

    // Warmup Routine
    checkCudaErrors(hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, size, size, size, &alpha, A_d, size, B_d, size, &beta, C_d_cub, size));

    // Let Warmup Routine Finish
    checkCudaErrors(hipDeviceSynchronize());

    // Record start point
    checkCudaErrors(hipEventRecord(start_cub, NULL));

    // Perform matrix multiplication
    checkCudaErrors(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, size, size, size, &alpha, B_d, size, A_d, size, &beta, C_d_cub, size));

    // Record end point
    checkCudaErrors(hipEventRecord(stop_cub, NULL));

    // Wait until the cuda event terminates
    checkCudaErrors(hipEventSynchronize(stop_cub));

    // Record elapsed time
    float ms_elapsed_cub;
    checkCudaErrors(hipEventElapsedTime(&ms_elapsed_cub, start_cub, stop_cub));

    // floating point operations per matmul
    double flops_cub = 2.0 * ((double) size) * ((double) size) * ((double) size);
    double gf_per_s_cub = flops_cub * 1.0e-9f /(ms_elapsed_cub / 1000.0f);

    // Print diagnostics
    printf("Cublas Performance Metrics: \n %.2f GFlops/s \n %.3f ms\n", gf_per_s_cub, ms_elapsed_cub);

    // Record start point
    checkCudaErrors(hipEventRecord(start_alg, NULL));

    // Run implemented algorithm
    mat_mul<<<grid_dim, block_dim>>>(A_d, B_d, C_d_alg, size);

    // Allow all threads to finish
    hipDeviceSynchronize();

    // Record stop point
    checkCudaErrors(hipEventRecord(stop_alg, NULL));

    // Wait until the cuda event terminates
    checkCudaErrors(hipEventSynchronize(stop_alg));

    // Record elapsed time
    float ms_elapsed_alg;
    checkCudaErrors(hipEventElapsedTime(&ms_elapsed_alg, start_alg, stop_alg));

    // floating point operations per matmul
    double flops_alg = 2.0 * ((double) size) * ((double) size) * ((double) size);
    double gf_per_s_alg = flops_alg * 1.0e-9f /(ms_elapsed_alg / 1000.0f);

    // Print diagnostics
    printf("Shared Memory Cache Blocking Algorithm Performance Metrics: \n %.2f GFlops/s \n %.3f ms\n", gf_per_s_alg, ms_elapsed_alg);

    // Copy reuslt back to host memory
    device_to_host(C_d_cub, C_cub, size);
    device_to_host(C_d_alg, C_alg, size);

    // Check for errors
    compare_matrix(C_alg, C_cub, size, 1.0e-1f);

    // cleanup memory
    free(A);
    free(B);
    free(C_cub);
    free(C_alg);

    // cleanup events
    hipEventDestroy(start_cub);
    hipEventDestroy(stop_cub);
    hipEventDestroy(start_alg);
    hipEventDestroy(stop_alg);

    // cleanup device memory
    checkCudaErrors(hipFree(A_d));
    checkCudaErrors(hipFree(B_d));
    checkCudaErrors(hipFree(C_d_cub));
    checkCudaErrors(hipFree(C_d_alg));

    // destroy cublas handle
    checkCudaErrors(hipblasDestroy(handle));

}



int main(int argc, char * argv[]) {

    int size = std::stoi(argv[1]);
    run_test(size);
    return 0;
}